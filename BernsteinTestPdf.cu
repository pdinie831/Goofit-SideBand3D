#include "hip/hip_runtime.h"
#define MAX_THREADS_PER_BLOCK 512
#define MIN_BLOCKS_PER_MP     20
#include <goofit/PDFs/mypdf/BernsteinTestPdf.h>
#include <goofit/Variable.h>

//   __global__ void
//   __launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)

namespace GooFit {

/* a struct for storing double numbers */
// struct bernVal {
//   double bernFunc;
//   double bernIntg;
// };


 __device__ __thrust_forceinline__ fptype device_coeffbinomial_ber(fptype enne, fptype kappa){
 
        fptype factor=1.;
        for(fptype i = 1; i <=kappa; ++i) {
          factor *= (enne+1-i)/i; 
        }	 
 
        if (factor<=0 ){
	 printf("Error in BernsteinTestPdf coeffbinomial=> factor = %f enne=%f kappa=%f",factor,enne,kappa);
         return 0;
	} 
       return factor;
}
 __device__ __thrust_forceinline__ fptype  device_BernsteinTestkn_func(fptype x, fptype enne, fptype kappa){
 
   return device_coeffbinomial_ber(enne,kappa)*pow(x,kappa)*pow(1.0-x,enne-kappa);


}
 __device__ fptype  device_BernsteinTestkn_intg(fptype x, fptype enne, fptype kappa){
 
//  	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
//       printf("==================================================\n");
//       printf("==================================================\n");
//      }
//       struct bernVal results;
//       results.bernFunc = 0;
//       results.bernIntg = 0;
      if (x<0 || x>1 ){
       printf(" Error in BernsteinTestkn_intg  x=%5.15f out of range [0,1]\n",x);
       return 0.;
      }
//      if (kappa>enne) return 0;
//      bernkn *= pow(x,kappa) ;
//      bernkn *= pow(1.0-x,enne-kappa) ;
      fptype integbernkn = 0;
      fptype ifactni = 0;
      fptype ifactik = 0;
      
       for(fptype i = kappa; i <=enne ; ++i) {
// n!/(i!(n-i)!)
//         ifactni=1;
//         for(float l = 1; l <=i; ++l) {
//           ifactni *= (maxDegree+1-l)/l; 
//         }	 

        ifactni =  device_coeffbinomial_ber(enne,i);
// i!/(k!(i-k)!)
//         ifactik=1;
//         for(float l = 1; l <=k; ++l) {
//           ifactik *= (i+1-l)/l; 
//         }	
 
        ifactik =  device_coeffbinomial_ber(i,kappa);
//
//        bernkn      += ifactni*ifactik*pow(menuno,i-kappa)*pow(x, i) ;
        integbernkn += ifactni*ifactik*pow(-1.0,i-kappa)/(i+1);
//	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
//          printf("pow(x=%5.15f,i=%5.15f)=%5.15f\n",x,i,pow(x, i ));
//          printf("pow(-1=%5.15f,i-kappa)=%5.15f\n",menuno,pow(menuno,i-kappa));
//          printf("BernsteinTestkn=%5.15f integral=%5.15f ifactni=%5.15f ifactik=%5.15f \n",bernkn,integbernkn,ifactni,ifactik);
//          printf("BernsteinTestkn=%f integral=%f kappa=%f i=%f enne=%f ni=%f nk=%f\n",bernkn,integbernkn,kappa,i,enne,ifactni,ifactik);
//        }
       }

       if (integbernkn<=0 ){
//	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
         printf(" Error in BernsteinTestkn_intg x=%5.15f integral = %5.15f THREADIDX=%d BLOCKIDX=%d\n", x,kappa,enne,integbernkn,THREADIDX,BLOCKIDX);
//	}
       }
//       results.bernFunc = bernkn;
//       results.bernIntg = integbernkn;
       return integbernkn;
}
//=======================================================================================================================
 __device__ fptype  device_BernsteinTestkn_intgBin( fptype xLeft, fptype xRight, fptype enne, fptype kappa){
 
//  	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
//       printf("==================================================\n");
//       printf("==================================================\n");
//      }
      fptype integbernkn = 0.0;
      fptype ifactni = 0.0;
      fptype ifactik = 0.0;
      fptype powxL = pow(xLeft ,kappa+1) ;
      fptype powxR = pow(xRight,kappa+1) ;
      
//	if( THREADIDX==98 && BLOCKIDX==0){ printf("=========================\n");}
       for(fptype i = kappa; i <=enne ; ++i) {
// n!/(i!(n-i)!)

        ifactni =  device_coeffbinomial_ber(enne,i);
// i!/(k!(i-k)!)
 
        ifactik =  device_coeffbinomial_ber(i,kappa);
//
//        bernkn      += ifactni*ifactik*pow(menuno,i-kappa)*pow(x, i) ;
//        integbernkn += ifactni*ifactik*pow(-1.0,i-kappa)*(pow(xRight, i+1)-pow(xLeft,i+1))/(i+1);
         integbernkn += ifactni*ifactik*pow(-1.0,i-kappa)*(powxR-powxL)/(i+1);
	 powxL*=xLeft;
	 powxR*=xRight;
// 	if( THREADIDX==98 && BLOCKIDX==0){
// 	 printf("integbernkn loop => enne=%15.15f kappa=%15.15f integbernkn+ = %15.15f ifactni=%15.15f ifactnk=%15.15f i=%15.15f\n",enne,kappa,ifactni*ifactik*pow(-1.0,i-kappa)/(i+1),ifactni,ifactik,i);
//   	}
//	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
//          printf("pow(x=%5.15f,i=%5.15f)=%5.15f\n",x,i,pow(x, i ));
//          printf("pow(-1=%5.15f,i-kappa)=%5.15f\n",menuno,pow(menuno,i-kappa));
//          printf("EffiBernsteinTestkn=%5.15f integral=%5.15f ifactni=%5.15f ifactik=%5.15f \n",bernkn,integbernkn,ifactni,ifactik);
//          printf("EffiBernsteinTestkn=%f integral=%f kappa=%f i=%f enne=%f ni=%f nk=%f\n",bernkn,integbernkn,kappa,i,enne,ifactni,ifactik);
//        }
       }

       if (integbernkn<=0.0 ){
	if ((476 == THREADIDX) && (148 == BLOCKIDX)){
         printf(" Error in BernsteinTestkn_intgbin xLeft=%f xRight=%f kappa=%f enne=%f integral = %5.15f THREADIDX=%d BLOCKIDX=%d\n",xLeft,xRight,kappa,enne,integbernkn,THREADIDX,BLOCKIDX);
	}
        integbernkn=1.E-30;
       }
//       results.bernFunc = bernkn;
//       results.bernIntg = integbernkn;
// // 	if( THREADIDX==98 && BLOCKIDX==0){
// // 	 printf("integbernkn => enne=%15.15f kappa=%15.15f integbernkn = %15.15f \n",enne,kappa,integbernkn);
// //          printf("=========================\n");
// // 	}
       return integbernkn;
}
//
//
//
//
//=================================================================================================================================== 
//
//================ device_BernsteinTest ======================================
// 
//=================================================================================================================================== 
__device__ fptype device_BernsteinTest(fptype *evt, fptype *p, unsigned int *indices) {
    // Structure is nP lowestdegree c1 c2 c3 nO o1
     
     
//    struct bernVal bernknval;

    int numParams = (indices[0]) ;
    int maxDegree = (indices[1]);

    fptype x   = evt[(indices[2 + (indices[0])])];
    fptype ret = 0;
    fptype integret = 0;
//    fptype bernkn = 0;
//    fptype integbernkn = 0;
    int ipar=2;
//    fptype ifactni=1;
//    fptype ifactik=1;
    fptype xmin=p[(indices[numParams-1])];
    fptype xmax=p[(indices[numParams])];
    x=(x-xmin)/(xmax-xmin);
//     printf("BernsteinTestPdf => limit xmin= %f xmax= %f\n",xmin,xmax);
//     return 0;


//     for(int i = 2; i < numParams; ++i) {
//         ret += (p[(indices[i])]) * pow(x, lowestDegree + i - 2);
//     }
    
      float k;
//      float i;
      for(k = 0; k <=maxDegree; ++k) {
       if (ipar>numParams-1){
        printf("Error in BernsteinTestPdf => ipar=%d > numParams=%d\n",ipar,numParams);
        return 0;
       }
       ret      += (p[(indices[ipar])]) * device_BernsteinTestkn_func(x,maxDegree,k);
       integret += (p[(indices[ipar])]) * device_BernsteinTestkn_intg(x,maxDegree,k);
//       printf("BernsteinTestPdf => %f integral = %f k=%d numparam=%d par=%f\n",ret,integret,k,numParams,(p[(indices[ipar])]));
       ipar++;
      }
//       printf("BernsteinTestPdf => %f int = %f\n",ret,integret);
    if(ret<1.E-30)  return 1.E-30;
    return ret/integret/(xmax-xmin);
//    return 0.;
}

/* __device__ fptype device_OffsetBernsteinTest(fptype *evt, fptype *p, unsigned int *indices) {
    int numParams    = (indices[0]);
    int lowestDegree = (indices[1]);

    fptype x = evt[(indices[2 + numParams])];
    x -= (p[(indices[numParams])]);
    fptype ret = 0;

    for(int i = 2; i < numParams; ++i) {
        ret += (p[(indices[i])]) * pow(x, lowestDegree + i - 2);
    }

    return ret*ret;
}
*/
//=================================================================================================================================== 
//
//================ device_MultiBernsteinTest ======================================
// 
//=================================================================================================================================== 
__device__ fptype device_MultiBernsteinTest(fptype *evt, fptype *p, unsigned int *indices) {
//       if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//        printf("==================================================\n");
//       }
// //     struct bernVal bernknvalx;
//     struct bernVal bernknvaly;
//     struct bernVal bernknvalz;
    int numObservables = (indices[(indices[0]) + 1]);
    int maxDegree1      = (indices[1]);
    int maxDegree2      = (indices[2]);
    int maxDegree3      = (indices[3]);
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiBernsteinTest 0=%d 0+1=%d 0+2=%d 0+3=%d\n",indices[(indices[0])],indices[(indices[0])+1],indices[(indices[0])+2],indices[(indices[0])+3]);
//      printf("MultiBernsteinTest numObservables=%d maxDegree1=%d maxDegree2=%d maxDegree3=%d\n",numObservables,indices[1],indices[2],indices[3]);
//      }
    if (numObservables!=3) {
     printf("device_MultiBernsteinTest error: Max Number of Observables is = 3!!! numObservables = %d\n",numObservables);
     return -100;
    }
 
    fptype x    = (evt[(indices[2 + (indices[0]) ])]); // x, y, z...
    fptype y    = (evt[(indices[2 + (indices[0]) + 1])]); // x, y, z...
    fptype z    = (evt[(indices[2 + (indices[0]) + 2])]); // x, y, z...
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiBernsteinTest x=%5.15f y=%5.15f z=%5.15f %d %d %d\n",x,y,z,numObservables,indices[1],indices[2],indices[3]);
//      }
    fptype xmin = (p[(indices[4 ])]);
    fptype xdif = (p[(indices[5 ])])-(p[(indices[4 ])]);
    x=(x-xmin)/xdif;
    fptype ymin = (p[(indices[6])]);
    fptype ydif = (p[(indices[7])])-(p[(indices[6])]);
    y=(y-ymin)/ydif;
    fptype zmin = (p[(indices[8])]);
    fptype zdif = (p[(indices[9])])-(p[(indices[8])]);
    z=(z-zmin)/zdif;
    
//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	printf("MultiBernsteinTest xmin=%5.15f xmax = %5.15f\n",xmin,xdif);
// 	printf("MultiBernsteinTest ymin=%5.15f ymax = %5.15f\n",ymin,ydif); 
// 	printf("MultiBernsteinTest zmin=%5.15f zmax = %5.15f\n",zmin,zdif);
// 	printf("MultiBernsteinTest [0,1] x=%5.15f y=%5.15f z=%5.15f \n",x,y,z);
//        
//        }
       double sx[10]={0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
   
       double sy[10]={0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
   
       double sz[10]={0.,0.,0.,0.,0.,0.,0.,0.,0.,0.};
       
       int enne = max(maxDegree3,max(maxDegree1,maxDegree2));
      
       if ( enne>9) {
        printf("device_MultiBernsteinTest error: Max(Numbers of degree) > 9 = %d\n",enne);
        return 0.0;
       }
       
       
       sx[0]=1.0;
       sy[0]=1.0;
       sz[0]=1.0;
//       for( int i = 1; i <= 9 ; ++i){
       for( int i = 1; i <= enne ; ++i){
        sx[i]= sx[i-1]*(1.-x);
        sy[i]= sy[i-1]*(1.-y);
        sz[i]= sz[i-1]*(1.-z);
       }
       int ipar =4 + 2*numObservables;
       int ipa0 =ipar;
//       int kk = 0;
//       int ii = 0;
//       int jj = 0;
       fptype func =0.0;
       fptype bernknvalx = 0.0;
       fptype bernknvaly = 0.0;
       fptype bernknvalz = 0.0;
       fptype intg_1 =0.0;
       fptype tx = 1.;
       for(int i = 0; i <= maxDegree1 ; ++i) {
         bernknvalx =  device_coeffbinomial_ber(maxDegree1,i)*tx*sx[maxDegree1-i];
//       jj = 0;
         fptype ty = 1.;
         for(int j = 0; j <= maxDegree2 ; ++j) {
	  bernknvaly =  device_coeffbinomial_ber(maxDegree2,j)*ty*sy[maxDegree2-j];
//	  std::cout<<"func = par["<<ipar<<"]*x^"<<kk<<"*y^"<<jj<<std::endl;
//          ii = 0;
          fptype tz = 1.;
          for(int k = 0; k <= maxDegree3 ; ++k) {
// 	   fptype bernknvalx =  device_coeffbinomial_ber(maxDegree1,i)*tx*pow(1.0-x,maxDegree1-i);
// 	   fptype bernknvaly =  device_coeffbinomial_ber(maxDegree2,j)*ty*pow(1.0-y,maxDegree2-j);
//  	   fptype bernknvalz =  device_coeffbinomial_ber(maxDegree3,k)*tz*pow(1.0-z,maxDegree3-k);
	   if (k==0) ipa0=ipar;
	   bernknvalz =  device_coeffbinomial_ber(maxDegree3,k)*tz*sz[maxDegree3-k];
//	   std::cout<<"func = par["<<ipar<<"]*x^"<<ii<"*y^"<<jj<<"*z^"<<kk<<std::endl;
//        	   fptype bernknvalx =  device_BernsteinTestkn_func(x,maxDegree1,i);
//     	   fptype bernknvaly =  device_BernsteinTestkn_func(y,maxDegree2,j);
//     	   fptype bernknvalz =  device_BernsteinTestkn_func(z,maxDegree3,k);
// 	   fptype bernknintx =  device_BernsteinTestkn_intg(x,maxDegree1,i);
// 	   fptype bernkninty =  device_BernsteinTestkn_intg(y,maxDegree2,j);
// 	   fptype bernknintz =  device_BernsteinTestkn_intg(z,maxDegree3,k);
//            func +=(p[(indices[ipar])])*bernknvalx*bernknvaly*bernknvalz;
//            intg +=(p[(indices[ipar])])*bernknintx*bernkninty*bernknintz;
 	   if(k==maxDegree3){
            func   +=(p[(indices[ipa0])])*bernknvalx*bernknvaly*bernknvalz;
            intg_1 +=(p[(indices[ipa0])]);
	   }else{
            func   +=(p[(indices[ipar])])*bernknvalx*bernknvaly*bernknvalz;
            intg_1 +=(p[(indices[ipar])]);
	    ipar++;
	   } 
// 	    if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  	     printf("MultiBernsteinTest  par = %f       \n",(p[(indices[ipar])]));
// 	     printf("MultiBernsteinTest  par = %f       B_(%d,%d,%d) = %f intg=%f\n",(p[(indices[ipar])]),ii,jj,kk,bernknvalx,bernknintx);
// 	    } 

//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	printf("MultiBernsteinTest MaxDegree=%d coefficient = %f   number = %d\n",maxDegree,(p[(indices[ipar])]),ipar-2-2*numObservables);
//        } 
	   
//           ii = (jj+kk+ii<maxDegree?++ii:0);
	   tz*=z;
	  }
//          jj = (jj+kk+ii<maxDegree?++jj:0);
	  
	   ty*=y;
	 
         }
//         kk= (jj+kk+ii<maxDegree?++kk:0);
	   tx*=x;
       }
//       return  func*func;
//       return  func/(intg);
//      return  func/(intg)/xdif/ydif/zdif;
//      return  func/(intg)/xdif/ydif/zdif;
      
// okkio se commenti qui sotto, cosi' non e' normalizzato!!!!!!!!!!!!!! 
      intg_1 = (1.0+maxDegree1)*(1.0+maxDegree2)*(1.0+maxDegree3)/intg_1;     
      intg_1 = intg_1/xdif*ydif*zdif;
      func=func*intg_1;
      if(func<1.E-30)  return 1.E-30;
      return  func;
 }
//
//=================================================================================================================================== 
//
//================ device_MultiBinBernsteinTest ======================================
// 
//=================================================================================================================================== 
__device__ fptype device_MultiBinBernsteinTest(fptype *evt, fptype *p, unsigned int *indices) {
//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//         printf("==================================================\n");
//        }
// //     struct bernVal bernknvalx;
//     struct bernVal bernknvaly;
//     struct bernVal bernknvalz;
    int numObservables  = (indices[(indices[0]) + 1]);
    int maxDegree1      = (indices[1]);
    int maxDegree2      = (indices[2]);
    int maxDegree3      = (indices[3]);
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiEffiBernsteinTest 0=%d 0+1=%d 0+2=%d 0+3=%d\n",indices[(indices[0])],indices[(indices[0])+1],indices[(indices[0])+2],indices[(indices[0])+3]);
//      printf("MultiEffiBernsteinTest numObservables=%d maxDegree1=%d maxDegree2=%d maxDegree3=%d\n",numObservables,indices[1],indices[2],indices[3]);
//      }
    if ( (numObservables)!=3) {
     printf("device_MultiBinBernsteinTest error: Max Number of Observables is = 3!!! numObservables = %d\n",numObservables);
     return -100;
    }
 
    fptype x    = (evt[(indices[2 + (indices[0]) ])]); // x, y, z...
    fptype y    = (evt[(indices[2 + (indices[0]) + 1])]); // x, y, z...
    fptype z    = (evt[(indices[2 + (indices[0]) + 2])]); // x, y, z...

    fptype xBinw =(p[(indices[10])]);
    fptype yBinw =(p[(indices[11])]);
    fptype zBinw =(p[(indices[12])]);
//    fptype zBinw =atan2(0.0,-1.0)/5.;
//    fptype zBinw =0.02;

    fptype xmin = (p[(indices[4 ])]);
    fptype xdif = (p[(indices[5 ])])-(p[(indices[4 ])]);

    fptype xLeft  = ((x-xBinw/2.)-xmin)/xdif;
    fptype xRight = ((x+xBinw/2.)-xmin)/xdif;
//    x=(x-xmin)/xdif;
    fptype ymin = (p[(indices[6])]);
    fptype ydif = (p[(indices[7])])-(p[(indices[6])]);
    fptype yLeft  = ((y-yBinw/2.)-ymin)/ydif;
    fptype yRight = ((y+yBinw/2.)-ymin)/ydif;
//    y=(y-ymin)/ydif;
    fptype zmin = (p[(indices[8])]);
    fptype zdif = (p[(indices[9])])-(p[(indices[8])]);
    fptype zLeft  = ((z-zBinw/2.)-zmin)/zdif;
    fptype zRight = ((z+zBinw/2.)-zmin)/zdif;
//    z=(z-zmin)/zdif;
    
       int ipar =4 + 3*(numObservables);
       int ipa0 =ipar;
//       int kk = 0;
//       int ii = 0;
//       int jj = 0;
       fptype ret   =0;
//       fptype intg =0;
       for(int i = 0; i <= maxDegree1 ; ++i) {
//       jj = 0;
         for(int j = 0; j <= maxDegree2 ; ++j) {
//	  std::cout<<"func = par["<<ipar<<"]*x^"<<kk<<"*y^"<<jj<<std::endl;
//          ii = 0;
          for(int k = 0; k <= maxDegree3 ; ++k) {

	   if (k==0) ipa0=ipar;
           fptype bernknintgbinx = device_BernsteinTestkn_intgBin(xLeft,xRight,maxDegree1,i);
           fptype bernknintgbiny = device_BernsteinTestkn_intgBin(yLeft,yRight,maxDegree2,j);
           fptype bernknintgbinz = device_BernsteinTestkn_intgBin(zLeft,zRight,maxDegree3,k);
	   if(k==maxDegree3){
            ret   +=(p[(indices[ipa0])])*bernknintgbinx*bernknintgbiny*bernknintgbinz;
	   }else{
            ret   +=(p[(indices[ipar])])*bernknintgbinx*bernknintgbiny*bernknintgbinz;
	    ipar++;
	   } 
//	   
//           ii = (jj+kk+ii<maxDegree?++ii:0);
	  }
//          jj = (jj+kk+ii<maxDegree?++jj:0);
	  
	 
         }
//         kk= (jj+kk+ii<maxDegree?++kk:0);
       }
    ret=ret/(xBinw*yBinw*zBinw);
    if(ret<1.E-30) ret = 1.E-30;

   return ret;

 }
//=================================================================================================================================== 
//
//================ device_MultiBinBernsteinWrongTest ======================================
// 
//=================================================================================================================================== 
__device__ fptype device_MultiBinBernsteinWrongTest(fptype *evt, fptype *p, unsigned int *indices) {
//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//         printf("==================================================\n");
//        }
// //     struct bernVal bernknvalx;
//     struct bernVal bernknvaly;
//     struct bernVal bernknvalz;
    int numObservables  = (indices[(indices[0]) + 1]);
    int maxDegree1      = (indices[1]);
    int maxDegree2      = (indices[2]);
    int maxDegree3      = (indices[3]);
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiEffiBernsteinTest 0=%d 0+1=%d 0+2=%d 0+3=%d\n",indices[(indices[0])],indices[(indices[0])+1],indices[(indices[0])+2],indices[(indices[0])+3]);
//      printf("MultiEffiBernsteinTest numObservables=%d maxDegree1=%d maxDegree2=%d maxDegree3=%d\n",numObservables,indices[1],indices[2],indices[3]);
//      }
    if ( (numObservables)!=3) {
     printf("device_MultiBinBernsteinWrongTest error: Max Number of Observables is = 3!!! numObservables = %d\n",numObservables);
     return -100;
    }
 
    fptype x    = (evt[(indices[2 + (indices[0]) ])]); // x, y, z...
    fptype y    = (evt[(indices[2 + (indices[0]) + 1])]); // x, y, z...
    fptype z    = fabs(evt[(indices[2 + (indices[0]) + 2])]); // x, y, z...

    fptype xBinw =(p[(indices[10])]);
    fptype yBinw =(p[(indices[11])]);
    fptype zBinw =(p[(indices[12])]);
//    fptype zBinw =atan2(0.0,-1.0)/5.;
//    fptype zBinw =0.02;

    fptype xmin = (p[(indices[4 ])]);
    fptype xdif = (p[(indices[5 ])])-(p[(indices[4 ])]);

    fptype xLeft  = ((x-xBinw/2.)-xmin)/xdif;
    fptype xRight = ((x+xBinw/2.)-xmin)/xdif;
//    x=(x-xmin)/xdif;
    fptype ymin = (p[(indices[6])]);
    fptype ydif = (p[(indices[7])])-(p[(indices[6])]);
    fptype yLeft  = ((y-yBinw/2.)-ymin)/ydif;
    fptype yRight = ((y+yBinw/2.)-ymin)/ydif;
//    y=(y-ymin)/ydif;
    fptype zmin = (p[(indices[8])]);
    fptype zdif = (p[(indices[9])])-(p[(indices[8])]);
    fptype zLeft  = ((z-zBinw/2.)-zmin)/zdif;
    fptype zRight = ((z+zBinw/2.)-zmin)/zdif;
//    z=(z-zmin)/zdif;
    
       int ipar =4 + 3*(numObservables);
       int ipa0 = ipar;
//       int kk = 0;
//       int ii = 0;
//       int jj = 0;
       fptype ret   =0;
//       fptype intg =0;
       for(int i = 0; i <= maxDegree1 ; ++i) {
//       jj = 0;
         for(int j = 0; j <= maxDegree2 ; ++j) {
//	  std::cout<<"func = par["<<ipar<<"]*x^"<<kk<<"*y^"<<jj<<std::endl;
//          ii = 0;
          for(int k = 0; k <= maxDegree3 ; ++k) {

	   if (k==0) ipa0=ipar;
           fptype bernknintgbinx = device_BernsteinTestkn_intgBin(xLeft,xRight,maxDegree1,i);
           fptype bernknintgbiny = device_BernsteinTestkn_intgBin(yLeft,yRight,maxDegree2,j);
           fptype bernknintgbinz = device_BernsteinTestkn_intgBin(zLeft,zRight,maxDegree3,k);
	   if(k==maxDegree3){
            ret   +=(p[(indices[ipa0])])*bernknintgbinx*bernknintgbiny*bernknintgbinz;
	   }else{
            ret   +=(p[(indices[ipar])])*bernknintgbinx*bernknintgbiny*bernknintgbinz;
	    ipar++;
	   } 
//           ii = (jj+kk+ii<maxDegree?++ii:0);
	  }
//          jj = (jj+kk+ii<maxDegree?++jj:0);
	  
	 
         }
//         kk= (jj+kk+ii<maxDegree?++kk:0);
       }
    if(ret<1.E-30) ret = 1.E-30;
 //   ret=ret/(xBinw*yBinw*zBinw);

   return ret;

 }
// device_MultiBinBernsteinWrongTest
//=================================================================================================================================== 
//
//================ device_MultiAdaptBernsteinTest ======================================
// 
//=================================================================================================================================== 
__device__ fptype device_MultiAdaptBernsteinTest(fptype *evt, fptype *p, unsigned int *indices) {
//         if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	         printf("==================================================\n");
// 		 
//         }
    int numObservables  = (indices[(indices[0]) + 1]);
    int maxDegree1      = (indices[1]);
    int maxDegree2      = (indices[2]);
    int maxDegree3      = (indices[3]);
    if ( (numObservables-3)!=3) {
     printf("device_MultiAdaptBernsteinTest error: Max Number of Observables is = 3!!! numObservables = %d\n",numObservables-3);
     return 0.0;
    }
 
    fptype x     = (evt[(indices[2 + (indices[0])    ])]); // x, y, z...
    fptype y     = (evt[(indices[2 + (indices[0]) + 1])]); // x, y, z...
    fptype z     = (evt[(indices[2 + (indices[0]) + 2])]); // x, y, z...
//     fptype reco  = (evt[(indices[2 + (indices[0]) + 3])]); // x, y, z...
//     fptype gene  = (evt[(indices[2 + (indices[0]) + 4])]); // x, y, z...
     fptype xBinw = (evt[(indices[2 + (indices[0]) + 3])]); // x, y, z...
     fptype yBinw = (evt[(indices[2 + (indices[0]) + 4])]); // x, y, z...
     fptype zBinw = (evt[(indices[2 + (indices[0]) + 5])]); // x, y, z...
//    fptype xBinw = 2./25.; // x, y, z...
//    fptype yBinw = 2./25.; // x, y, z...
//    fptype zBinw = 2*3.1415926535/25.; // x, y, z...
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      if ((THREADIDX==0) && (BLOCKIDX==0)){
//        printf("MultiEffiBernsteinTest x=%f y=%f z=%f xBinw=%f yBinw=%f zBinw=%f\n",x,y,z,xBinw,yBinw,zBinw);
//        printf("MultiEffiBernsteinTest numObservables=%d maxDegree1=%d maxDegree2=%d maxDegree3=%d\n",numObservables,indices[1],indices[2],indices[3]);
//        }
//     if (gene < reco || gene<0.0) {
//         printf("device_MultiAdaptBernsteinTest error: gene=%f < reco=%f THREADIDX==%d BLOCKIDX==%d\n",gene,reco,THREADIDX,BLOCKIDX);
//         return 0;
//     }
//    fptype nmax = 0;
    
// non se po' fa'?    if (reco==0.0) return exp( -1.0E30);
//    if (gene> nmax) nmax = gene;
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiEffiBernsteinTest x=%5.15f y=%5.15f z=%5.15f %d %d %d\n",x,y,z,numObservables,indices[1],indices[2],indices[3]);
//      }    

//     fptype xBinw =(p[(indices[10])]);
//     fptype yBinw =(p[(indices[11])]);
//     fptype zBinw =(p[(indices[12])]);
//    fptype zBinw =atan2(0.0,-1.0)/5.;
//    fptype zBinw =0.02;

    fptype xmin = (p[(indices[4 ])]);
    fptype xdif = (p[(indices[5 ])])-(p[(indices[4 ])]);

    fptype xLeft  = ((x-xBinw/2.)-xmin)/xdif;
    fptype xRight = ((x+xBinw/2.)-xmin)/xdif;
//    x=(x-xmin)/xdif;
    fptype ymin = (p[(indices[6])]);
    fptype ydif = (p[(indices[7])])-(p[(indices[6])]);
    fptype yLeft  = ((y-yBinw/2.)-ymin)/ydif;
    fptype yRight = ((y+yBinw/2.)-ymin)/ydif;
//    y=(y-ymin)/ydif;
    fptype zmin = (p[(indices[8])]);
    fptype zdif = (p[(indices[9])])-(p[(indices[8])]);
    fptype zLeft  = ((z-zBinw/2.)-zmin)/zdif;
    fptype zRight = ((z+zBinw/2.)-zmin)/zdif;
//    z=(z-zmin)/zdif;
    
//  	if ( (47 == THREADIDX) && (0 == BLOCKIDX)){
//         printf("==================================================\n");
// //   printf("EffiBernsteinTestPdf THREADIDX==%d BLOCKIDX==%d\n",THREADIDX,BLOCKIDX);
//   	 printf("MultiEffiBernsteinTest x=%5.15f y=%5.15f z=%5.15f\n",x,y,z);
// //  	 printf("MultiEffiBernsteinTest xmin=%5.15f xdif = %5.15f\n",xmin,xdif);
// //  	 printf("MultiEffiBernsteinTest ymin=%5.15f ydif = %5.15f\n",ymin,ydif);
//   	 printf("MultiEffiBernsteinTest zmin=%5.15f zdif = %5.15f\n",zmin,zdif);
// //  	 printf("MultiEffiBernsteinTest xLeft=%5.15f xRight = %5.15f\n",xLeft,xRight);
// //  	 printf("MultiEffiBernsteinTest yLeft=%5.15f yRight = %5.15f\n",yLeft,yRight);
//   	 printf("MultiEffiBernsteinTest zLeft=%5.15f zRight = %5.15f\n",zLeft,zRight);
//  	}
       int ipar = 4 + numObservables*2; 
       int ipa0 = ipar;
       //conteggio indici: gli interi [numero Obs + 3 gradi del poly (maxDeg1,2,3) = 4] + num obs(vars+bins) + num limiti (6)
//       int kk = 0;
//       int ii = 0;
//       int jj = 0;
       fptype ret  =0.0;
       fptype intg_1 =0.;
//       fptype mu   =0.0;
//       fptype intg_1 =0;
       for(int i = 0; i <= maxDegree1 ; ++i) {
//       jj = 0;
         for(int j = 0; j <= maxDegree2 ; ++j) {
//	  std::cout<<"func = par["<<ipar<<"]*x^"<<kk<<"*y^"<<jj<<std::endl;
//          ii = 0;
          for(int k = 0; k <= maxDegree3 ; ++k) {
//	   std::cout<<"func = par["<<ipar<<"]*x^"<<ii<<"*y^"<<jj<<"*z^"<<kk<<std::endl;
//  	   fptype bernknvalx =  device_coeffbinomial_ber(maxDegree,ii)*pow(x,ii)*pow(1.0-x,maxDegree-ii);
//  	   fptype bernknvaly =  device_coeffbinomial_ber(maxDegree,jj)*pow(x,jj)*pow(1.0-x,maxDegree-jj);
//  	   fptype bernknvalz =  device_coeffbinomial_ber(maxDegree,kk)*pow(x,kk)*pow(1.0-x,maxDegree-kk);
//         fptype bernknvalx =  device_EffiBernsteinTestkn_func(x,maxDegree1,i);
//     	   fptype bernknvaly =  device_EffiBernsteinTestkn_func(y,maxDegree2,j);
//     	   fptype bernknvalz =  device_EffiBernsteinTestkn_func(z,maxDegree3,k);
// 	   fptype bernknintx =  device_EffiBernsteinTestkn_intg(maxDegree1,i);
// 	   fptype bernkninty =  device_EffiBernsteinTestkn_intg(maxDegree2,j);
// 	   fptype bernknintz =  device_EffiBernsteinTestkn_intg(maxDegree3,k);
//            func +=(p[(indices[ipar])])*bernknvalx*bernknvaly*bernknvalz;
//            intg +=(p[(indices[ipar])])*bernknintx*bernkninty*bernknintz;

	   if (k==0) ipa0=ipar;
           fptype bernknintgbinx = device_BernsteinTestkn_intgBin(xLeft,xRight,maxDegree1,i);
           fptype bernknintgbiny = device_BernsteinTestkn_intgBin(yLeft,yRight,maxDegree2,j);
           fptype bernknintgbinz = device_BernsteinTestkn_intgBin(zLeft,zRight,maxDegree3,k);
//           mu   +=(p[(indices[ipar])])*bernknintgbinx/(xBinw);
//           mu   +=(p[(indices[ipar])])*bernknintgbiny/(yBinw);
//           mu   +=(p[(indices[ipar])])*bernknintgbinz/(zBinw);
//           mu   +=(p[(indices[ipar])])*bernknintgbinx*bernknintgbiny*bernknintgbinz/(xBinw*yBinw*zBinw);
 	   if(k==maxDegree3){
            ret    +=(p[(indices[ipa0])])*bernknintgbinx*bernknintgbiny*bernknintgbinz;
            intg_1 +=(p[(indices[ipa0])]);
 	   }else{
            ret    +=(p[(indices[ipar])])*bernknintgbinx*bernknintgbiny*bernknintgbinz;
            intg_1 +=(p[(indices[ipar])]);
	    ipar++;
	   } 
//           intg_1 +=(p[(indices[ipar])]);
//  	if ( (47 == THREADIDX) && (0 == BLOCKIDX)){
// //  	 printf("MultiEffiBernsteinTest bernknintgbinx=%5.15f\n",bernknintgbinx);
// //  	 printf("MultiEffiBernsteinTest bernknintgbiny=%5.15f\n",bernknintgbiny);
//   	 printf("MultiEffiBernsteinTest bernknintgbinz=%5.15f\n",bernknintgbinz);
// 	} 
//           intg +=(p[(indices[ipar])]);
//           intg +=(p[(indices[ipar])])*bernknintx*bernkninty*bernknintz;
//   	    if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//    	     printf("MultiEffiBernsteinTest  par = %f   ipar=%d    \n",(p[(indices[ipar])]),ipar);
// // // 	     printf("MultiEffiBernsteinTest  par = %f       B_(%d,%d,%d) = %f intg=%f\n",(p[(indices[ipar])]),ii,jj,kk,bernknvalx,bernknintx);
//   	    } 

//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	printf("MultiEffiBernsteinTest MaxDegree=%d coefficient = %f   number = %d\n",maxDegree,(p[(indices[ipar])]),ipar-2-2*numObservables);
//        } 
	   
//           ii = (jj+kk+ii<maxDegree?++ii:0);
	  }
//          jj = (jj+kk+ii<maxDegree?++jj:0);
	  
	 
         }
//         kk= (jj+kk+ii<maxDegree?++kk:0);
       }
      if(ret<1.E-30) ret = 1.E-30;
//       intg_1 = (1.0+maxDegree1)*(1.0+maxDegree2)*(1.0+maxDegree3)/intg_1;
//       intg_1 = intg_1/(xdif*ydif*zdif);
//       ret=ret*intg_1/(xBinw*yBinw*zBinw);
//      ret=ret/(xBinw*yBinw*zBinw);// se commenti questo non hai piu' la densita' di probabilita' sul bin, ma (direttamente) la probabilita'
      intg_1 = (1.0+maxDegree1)*(1.0+maxDegree2)*(1.0+maxDegree3)/intg_1;
      ret = ret*intg_1;
      return ret;
 }

__device__ device_function_ptr ptr_to_BernsteinTest		    = device_BernsteinTest;
__device__ device_function_ptr ptr_to_MultiBernsteinTest	    = device_MultiBernsteinTest;
__device__ device_function_ptr ptr_to_MultiBinBernsteinTest	    = device_MultiBinBernsteinTest;
__device__ device_function_ptr ptr_to_MultiBinBernsteinWrongTest    = device_MultiBinBernsteinWrongTest;
__device__ device_function_ptr ptr_to_MultiAdaptBernsteinTest       = device_MultiAdaptBernsteinTest;

// Constructor for single-variate BernsteinTest, with optional zero point.
// __host__ BernsteinTestPdf::BernsteinTestPdf(std::string n, Observable _x, std::vector<Variable> weights, unsigned int lowestDegree)
//     : GooPdf(n, _x) {
//     std::vector<unsigned int> pindices;
//     pindices.push_back(lowestDegree);
// 
//     for(auto &weight : weights) {
//         pindices.push_back(registerParameter(weight));
//     }
// 
//     GET_FUNCTION_ADDR(ptr_to_BernsteinTest);
// 
//     initialize(pindices);
// }

//=========================================================== 
//Constructor for single-variate BernsteinTest.
//=========================================================== 
__host__ BernsteinTestPdf::BernsteinTestPdf(std::string n, Observable _x, std::vector<Variable> weights,std::vector<Variable> limits, unsigned int maxDegree)
    : GooPdf(n, _x) {
    std::vector<unsigned int> pindices;
    pindices.push_back(maxDegree);

    for(auto &weight : weights) {
        pindices.push_back(registerParameter(weight));
    }
    for(auto &limit : limits) {
        pindices.push_back(registerParameter(limit));
    }

     GET_FUNCTION_ADDR(ptr_to_BernsteinTest);
//    GET_FUNCTION_ADDR(ptr_to_OffsetBernsteinTest);

    initialize(pindices);
}
//=========================================================== 
// Constructor for multivariate BernsteinTest.
//=========================================================== 
 __host__ BernsteinTestPdf::BernsteinTestPdf(std::string n,
				       std::vector<Observable> obses,
				       std::vector<Variable> coeffs,
				       std::vector<Variable> limits,
				       unsigned int maxDegree1,
				       unsigned int maxDegree2,
				       unsigned int maxDegree3 )
        : GooPdf(n) {
     unsigned int numParameters = 1;
 
      size_t limit = 0;
      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("hipLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceSetLimit(hipLimitStackSize, 2*limit);
      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("hipLimitStackSize: %u\n", (unsigned)limit);
      printf("GooFit::Multivariate BernsteinTest, num obs:%d\n",obses.size());
     // For 1 observable, equal to n = maxDegree + 1.
     // For two, n*(n+1)/2, ie triangular number. This generalises:
     // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
     // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
     // ...
     for(unsigned int i = 0; i < obses.size(); ++i) {
	 registerObservable(obses[i]);
//	 numParameters *= (maxDegree + 1 + i);
     }
//  
//      for(int i = observables.size(); i > 1; --i)
// 	 numParameters /= i;
 
//     int j=1;
//     numParameters = pow((maxDegree+1),coeffs.size());
     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3);
//     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3+1);
     while(numParameters > coeffs.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));
 
	 coeffs.emplace_back(varName, 10.,0.00001,0.,500.);
 
	 std::cout << "Warning: " << getName() << " created dummy variable " << varName
		   << "  to account for all terms.\n";
     }
 
     while(limits.size() < 2*obses.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_limits_%i", getName().c_str(), static_cast<int>(limits.size()));
	 limits.emplace_back(varName, 0);
     }
 
     std::vector<unsigned int> pindices;
     pindices.push_back(maxDegree1);
     pindices.push_back(maxDegree2);
     pindices.push_back(maxDegree3);
 
     for(auto &limit : limits) {
	 pindices.push_back(registerParameter(limit));
     }
 
     for(auto &coeff : coeffs) {
	 pindices.push_back(registerParameter(coeff));
     }
 
     GET_FUNCTION_ADDR(ptr_to_MultiBernsteinTest);
     initialize(pindices);
 }
//=========================================================== 
// Constructor for multivariate BIN BernsteinTest.
//=========================================================== 
 __host__ BernsteinTestPdf::BernsteinTestPdf(std::string n,
				     std::vector<Observable> obses,
				     std::vector<Variable> coeffs,
				     std::vector<Variable> limits,
				     std::vector<Variable> binws,
				     unsigned int maxDegree1,
				     unsigned int maxDegree2,
				     unsigned int maxDegree3 )
        : GooPdf(n) {
     unsigned int numParameters = 1;
      size_t limit = 0;
      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("hipLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceSetLimit(hipLimitStackSize, 2*limit);
      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("hipLimitStackSize: %u\n", (unsigned)limit);
      printf("GooFit::Multivariate Bin BernsteinTest, num obs:%d\n",obses.size());
 
     // For 1 observable, equal to n = maxDegree + 1.
     // For two, n*(n+1)/2, ie triangular number. This generalises:
     // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
     // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
     // ...
     for(unsigned int i = 0; i < obses.size(); ++i) {
	 registerObservable(obses[i]);
//	 numParameters *= (maxDegree + 1 + i);
     }
//  
//      for(int i = observables.size(); i > 1; --i)
// 	 numParameters /= i;
 
//     int j=1;
//     numParameters = pow((maxDegree+1),coeffs.size());
     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3);
//     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3+1);
     while(numParameters > coeffs.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));
 
	 coeffs.emplace_back(varName, 10.,0.00001,0.,500.);
 
	 std::cout << "Warning: " << getName() << " created dummy variable " << varName
		   << "  to account for all terms.\n";
     }
 
     while(limits.size() < 2*(obses.size()-2)) {
	 char varName[100];
	 sprintf(varName, "%s_extra_limits_%i", getName().c_str(), static_cast<int>(limits.size()));
	 limits.emplace_back(varName, 0);
     }
 
     std::vector<unsigned int> pindices;
     pindices.push_back(maxDegree1);
     pindices.push_back(maxDegree2);
     pindices.push_back(maxDegree3);
 
     for(auto &limit : limits) {
	 pindices.push_back(registerParameter(limit));
     }
     for(auto &binw : binws) {
	 pindices.push_back(registerParameter(binw));
     }
 
     for(auto &coeff : coeffs) {
	 pindices.push_back(registerParameter(coeff));
     }
 
     GET_FUNCTION_ADDR(ptr_to_MultiBinBernsteinTest);
     initialize(pindices);
 }
 
//=========================================================== 
// Constructor for multivariate BIN BernsteinTest WRONG.
//=========================================================== 
 __host__ BernsteinTestPdf::BernsteinTestPdf(std::string n,
				     std::vector<Observable> obses,
				     std::vector<Variable>   coeffs,
				     std::vector<Variable>   limits,
				     std::vector<Variable>   binws,
				     unsigned int maxDegree1,
				     unsigned int maxDegree2,
				     unsigned int maxDegree3,
				     unsigned int dummy1,
				     unsigned int dummy2 )
        : GooPdf(n) {
     unsigned int numParameters = 1;
      size_t limit = 0;
      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("hipLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceSetLimit(hipLimitStackSize, 2*limit);
      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("hipLimitStackSize: %u\n", (unsigned)limit);
      printf("GooFit::Multivariate Bin Wrong BernsteinTest, num obs:%d\n",obses.size());
 
     // For 1 observable, equal to n = maxDegree + 1.
     // For two, n*(n+1)/2, ie triangular number. This generalises:
     // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
     // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
     // ...
     for(unsigned int i = 0; i < obses.size(); ++i) {
	 registerObservable(obses[i]);
//	 numParameters *= (maxDegree + 1 + i);
     }
//  
//      for(int i = observables.size(); i > 1; --i)
// 	 numParameters /= i;
 
//     int j=1;
//     numParameters = pow((maxDegree+1),coeffs.size());
     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3);
//     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3+1);
     while(numParameters > coeffs.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));
 
	 coeffs.emplace_back(varName, 10.,0.00001,0.,500.);
 
	 std::cout << "Warning: " << getName() << " created dummy variable " << varName
		   << "  to account for all terms.\n";
     }
 
     while(limits.size() < 2*(obses.size()-2)) {
	 char varName[100];
	 sprintf(varName, "%s_extra_limits_%i", getName().c_str(), static_cast<int>(limits.size()));
	 limits.emplace_back(varName, 0);
     }
 
     std::vector<unsigned int> pindices;
     pindices.push_back(maxDegree1);
     pindices.push_back(maxDegree2);
     pindices.push_back(maxDegree3);
 
     for(auto &limit : limits) {
	 pindices.push_back(registerParameter(limit));
     }
     for(auto &binw : binws) {
	 pindices.push_back(registerParameter(binw));
     }
 
     for(auto &coeff : coeffs) {
	 pindices.push_back(registerParameter(coeff));
     }
 
     GET_FUNCTION_ADDR(ptr_to_MultiBinBernsteinWrongTest);
     initialize(pindices);
 }
//=========================================================== 
// Constructor for multivariate Adaptive BernsteinTest.
//=========================================================== 
 __host__ BernsteinTestPdf::BernsteinTestPdf(std::string n,
				       std::vector<Observable> obses,
				       std::vector<Variable> coeffs,
				       std::vector<Variable> limits,
				       unsigned int maxDegree1,
				       unsigned int maxDegree2,
				       unsigned int maxDegree3,
				       unsigned int dummy )
        : GooPdf(n) {
     unsigned int numParameters = 1;
 //      size_t limit = 0;
//       hipDeviceGetLimit(&limit, hipLimitStackSize);
//       printf("hipLimitStackSize: %u\n", (unsigned)limit);
//       hipDeviceSetLimit(hipLimitStackSize, 2*limit);
//       hipDeviceGetLimit(&limit, hipLimitStackSize);
//       printf("hipLimitStackSize: %u\n", (unsigned)limit);
      printf("GooFit::Multivariate Adaptive BernsteinTest, num obs:%d\n",obses.size());
     // For 1 observable, equal to n = maxDegree + 1.
     // For two, n*(n+1)/2, ie triangular number. This generalises:
     // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
     // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
     // ...
     for(unsigned int i = 0; i < obses.size(); ++i) {
	 registerObservable(obses[i]);
//	 numParameters *= (maxDegree + 1 + i);
     }
//  
//      for(int i = observables.size(); i > 1; --i)
// 	 numParameters /= i;
 
//     int j=1;
//     numParameters = pow((maxDegree+1),coeffs.size());
     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3);
//     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3+1);
     while(numParameters > coeffs.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));
 
	 coeffs.emplace_back(varName, 10.,0.00001,0.,500.);
 
	 std::cout << "Warning: " << getName() << " created dummy variable " << varName
		   << "  to account for all terms.\n";
     }
 
     while(limits.size() < 2*obses.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_limits_%i", getName().c_str(), static_cast<int>(limits.size()));
	 limits.emplace_back(varName, 0);
     }
 
     std::vector<unsigned int> pindices;
     pindices.push_back(maxDegree1);
     pindices.push_back(maxDegree2);
     pindices.push_back(maxDegree3);
 
     for(auto &limit : limits) {
	 pindices.push_back(registerParameter(limit));
     }
 
     for(auto &coeff : coeffs) {
	 pindices.push_back(registerParameter(coeff));
     }
 
     GET_FUNCTION_ADDR(ptr_to_MultiAdaptBernsteinTest);
     initialize(pindices);
 }
 
//========================================================================
//======================================================================== 
 __host__ fptype BernsteinTestPdf::integrate(fptype lo, fptype hi) const {
       return 1.0;
 }

} // namespace GooFit
